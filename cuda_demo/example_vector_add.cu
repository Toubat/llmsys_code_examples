#include <hip/hip_runtime.h>

#include <iostream>
#include <sstream>
#include <fstream>

__global__ void VecAddKernel(int* A, int* B, int* C, int n) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i < n) {
    C[i] = A[i] + B[i];
  }
}


extern "C" {

void VecAddCPU(int* A, int* B, int* C, int n) {
  for(int i = 0; i < n; ++i) {
    C[i] = A[i] + B[i];
  }
}


void VecAddCUDA(int* Agpu, int* Bgpu, int* Cgpu, int n) {
  int threads_per_block = 256;
  int num_blocks = (n + threads_per_block - 1) / threads_per_block;
  VecAddKernel<<<num_blocks, threads_per_block>>>(Agpu, Bgpu, Cgpu, n);
}


void VecAddCUDA2(int* Acpu, int* Bcpu, int* Ccpu, int n) {
  int *dA, *dB, *dC;
  hipMalloc(&dA, n * sizeof(int));
  hipMalloc(&dB, n * sizeof(int));
  hipMalloc(&dC, n * sizeof(int));
  hipMemcpy(dA, Acpu, n * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(dB, Bcpu, n * sizeof(int), hipMemcpyHostToDevice);
  int threads_per_block = 256;
  int num_blocks = (n + threads_per_block - 1) / threads_per_block;
  VecAddKernel<<<num_blocks, threads_per_block>>>(dA, dB, dC, n);
  hipMemcpy(Ccpu, dC, n * sizeof(int), hipMemcpyDeviceToHost);
  hipFree(dA); 
  hipFree(dB); 
  hipFree(dC);
}

}